#include <simsense/core.h>
#include <simsense/camera.h>
#include <simsense/csct.h>
#include <simsense/cost.h>
#include <simsense/aggr.h>
#include <simsense/wta.h>
#include <simsense/lrcheck.h>
#include <simsense/filter.h>

#define gpuErrCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess) {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) { exit(code); }
   }
}

namespace simsense {

// Constructor without registration
DepthSensorEngine::DepthSensorEngine(
        uint32_t _rows, uint32_t _cols, float _focalLen, float _baselineLen, float _minDepth, float _maxDepth, uint64_t infraredNoiseSeed,
        float _speckleShape, float _speckleScale, float _gaussianMu, float _gaussianSigma, bool _rectified, uint8_t _censusWidth, uint8_t _censusHeight,
        uint32_t _maxDisp, uint8_t _bfWidth, uint8_t _bfHeight, uint8_t _p1, uint8_t _p2, uint8_t _uniqRatio, uint8_t _lrMaxDiff,
        uint8_t _mfSize, py::array_t<float> map_lx, py::array_t<float> map_ly, py::array_t<float> map_rx, py::array_t<float> map_ry
) {
    // Convert from python to C++
    Mat2d<float> mapLx = ndarray2Mat2d<float>(map_lx);
    Mat2d<float> mapLy = ndarray2Mat2d<float>(map_ly);
    Mat2d<float> mapRx = ndarray2Mat2d<float>(map_rx);
    Mat2d<float> mapRy = ndarray2Mat2d<float>(map_ry);

    // Create streams and free memory if necessary
    gpuErrCheck(hipStreamCreate(&stream1));
    gpuErrCheck(hipStreamCreate(&stream2));
    gpuErrCheck(hipStreamCreate(&stream3));

    // Intialize class variables
    speckleShape = _speckleShape;
    speckleScale = _speckleScale;
    gaussianMu = _gaussianMu;
    gaussianSigma = _gaussianSigma;
    censusWidth = _censusWidth;
    censusHeight = _censusHeight;
    bfWidth = _bfWidth;
    bfHeight = _bfHeight;
    p1 = _p1;
    p2 = _p2;
    uniqRatio = _uniqRatio;
    lrMaxDiff = _lrMaxDiff;
    mfSize = _mfSize;
    rows = _rows;
    cols = _cols;
    size = rows*cols;
    maxDisp = _maxDisp;
    int size3d = size*maxDisp;
    focalLen = _focalLen;
    baselineLen = _baselineLen;
    minDepth = _minDepth;
    maxDepth = _maxDepth;
    rectified = _rectified;
    registration = false;

    // Allocate GPU memory
    gpuErrCheck(hipMalloc((void **)&d_irNoiseStates0, sizeof(hiprandState_t)*size));
    gpuErrCheck(hipMalloc((void **)&d_irNoiseStates1, sizeof(hiprandState_t)*size));
    initInfraredNoise<<<(size + WARP_SIZE - 1) / WARP_SIZE, WARP_SIZE, 0, stream1>>>(
      static_cast<hiprandState_t *>(d_irNoiseStates0), infraredNoiseSeed, size);
    initInfraredNoise<<<(size + WARP_SIZE - 1) / WARP_SIZE, WARP_SIZE, 0, stream2>>>(
      static_cast<hiprandState_t *>(d_irNoiseStates1), infraredNoiseSeed + 1, size);

    gpuErrCheck(hipMalloc((void **)&d_mapLx, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_mapLy, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_mapRx, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_mapRy, sizeof(float)*size));
    gpuErrCheck(hipMemcpyAsync(d_mapLx, mapLx.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_mapLy, mapLy.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_mapRx, mapRx.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_mapRy, mapRy.data(), sizeof(float)*size, hipMemcpyHostToDevice));

    gpuErrCheck(hipMalloc((void **)&d_rawim0, sizeof(uint8_t)*size));
    gpuErrCheck(hipMalloc((void **)&d_rawim1, sizeof(uint8_t)*size));
    gpuErrCheck(hipMalloc((void **)&d_noisyim0, sizeof(uint8_t)*size));
    gpuErrCheck(hipMalloc((void **)&d_noisyim1, sizeof(uint8_t)*size));
    if (!rectified) {
        gpuErrCheck(hipMalloc((void **)&d_recim0, sizeof(uint8_t)*size));
        gpuErrCheck(hipMalloc((void **)&d_recim1, sizeof(uint8_t)*size));
    }

    gpuErrCheck(hipMalloc((void **)&d_census0, sizeof(uint32_t)*size));
    gpuErrCheck(hipMalloc((void **)&d_census1, sizeof(uint32_t)*size));

    if (bfWidth * bfHeight != 1) {
        gpuErrCheck(hipMalloc((void **)&d_rawcost, sizeof(cost_t)*size3d));
        gpuErrCheck(hipMalloc((void **)&d_hsum, sizeof(cost_t)*size3d));
    }
    gpuErrCheck(hipMalloc((void **)&d_cost, sizeof(cost_t)*size3d));

    gpuErrCheck(hipMalloc((void **)&d_L0, sizeof(cost_t)*size3d));
    gpuErrCheck(hipMalloc((void **)&d_L1, sizeof(cost_t)*size3d));
    gpuErrCheck(hipMalloc((void **)&d_L2, sizeof(cost_t)*size3d));
    gpuErrCheck(hipMalloc((void **)&d_LAll, sizeof(cost_t)*size3d));

    gpuErrCheck(hipMalloc((void **)&d_leftDisp, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_rightDisp, sizeof(uint16_t)*size));

    if (mfSize != 1) {
        gpuErrCheck(hipMalloc((void **)&d_filteredDisp, sizeof(float)*size));
    }

#ifdef DISP_ONLY
    h_disp = new float[size];
#else
    gpuErrCheck(hipMalloc((void **)&d_depth, sizeof(float)*size));
    h_depth = new float[size];
#endif

    gpuErrCheck(hipDeviceSynchronize());
}

// Constructor with registration
DepthSensorEngine::DepthSensorEngine(
        uint32_t _rows, uint32_t _cols, uint32_t _rgbRows, uint32_t _rgbCols, float _focalLen, float _baselineLen, float _minDepth,
        float _maxDepth, uint64_t infraredNoiseSeed, float _speckleShape, float _speckleScale, float _gaussianMu, float _gaussianSigma,
        bool _rectified, uint8_t _censusWidth, uint8_t _censusHeight, uint32_t _maxDisp, uint8_t _bfWidth, uint8_t _bfHeight, uint8_t _p1,
        uint8_t _p2, uint8_t _uniqRatio, uint8_t _lrMaxDiff, uint8_t _mfSize, py::array_t<float> map_lx, py::array_t<float> map_ly,
        py::array_t<float> map_rx, py::array_t<float> map_ry, py::array_t<float> _a1, py::array_t<float> _a2, py::array_t<float> _a3,
        float _b1, float _b2, float _b3, bool _dilation
) {
    // Convert from python to C++
    Mat2d<float> mapLx = ndarray2Mat2d<float>(map_lx);
    Mat2d<float> mapLy = ndarray2Mat2d<float>(map_ly);
    Mat2d<float> mapRx = ndarray2Mat2d<float>(map_rx);
    Mat2d<float> mapRy = ndarray2Mat2d<float>(map_ry);
    Mat2d<float> a1 = ndarray2Mat2d<float>(_a1);
    Mat2d<float> a2 = ndarray2Mat2d<float>(_a2);
    Mat2d<float> a3 = ndarray2Mat2d<float>(_a3);

    // Create streams and free memory if necessary
    gpuErrCheck(hipStreamCreate(&stream1));
    gpuErrCheck(hipStreamCreate(&stream2));
    gpuErrCheck(hipStreamCreate(&stream3));

    // Intialize class variables
    speckleShape = _speckleShape;
    speckleScale = _speckleScale;
    gaussianMu = _gaussianMu;
    gaussianSigma = _gaussianSigma;
    censusWidth = _censusWidth;
    censusHeight = _censusHeight;
    bfWidth = _bfWidth;
    bfHeight = _bfHeight;
    p1 = _p1;
    p2 = _p2;
    uniqRatio = _uniqRatio;
    lrMaxDiff = _lrMaxDiff;
    mfSize = _mfSize;
    rows = _rows;
    cols = _cols;
    size = rows*cols;
    maxDisp = _maxDisp;
    int size3d = size*maxDisp;
    focalLen = _focalLen;
    baselineLen = _baselineLen;
    minDepth = _minDepth;
    maxDepth = _maxDepth;
    rectified = _rectified;
    registration = true;
    dilation = _dilation;
    rgbRows = _rgbRows;
    rgbCols = _rgbCols;
    rgbSize = rgbRows*rgbCols;
    b1 = _b1;
    b2 = _b2;
    b3 = _b3;

    // Allocate GPU memory
    gpuErrCheck(hipMalloc((void **)&d_irNoiseStates0, sizeof(hiprandState_t)*size));
    gpuErrCheck(hipMalloc((void **)&d_irNoiseStates1, sizeof(hiprandState_t)*size));
    initInfraredNoise<<<(size + WARP_SIZE - 1) / WARP_SIZE, WARP_SIZE, 0, stream1>>>(
      static_cast<hiprandState_t *>(d_irNoiseStates0), infraredNoiseSeed, size);
    initInfraredNoise<<<(size + WARP_SIZE - 1) / WARP_SIZE, WARP_SIZE, 0, stream2>>>(
      static_cast<hiprandState_t *>(d_irNoiseStates1), infraredNoiseSeed + 1, size);

    gpuErrCheck(hipMalloc((void **)&d_mapLx, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_mapLy, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_mapRx, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_mapRy, sizeof(float)*size));
    gpuErrCheck(hipMemcpyAsync(d_mapLx, mapLx.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_mapLy, mapLy.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_mapRx, mapRx.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_mapRy, mapRy.data(), sizeof(float)*size, hipMemcpyHostToDevice));

    gpuErrCheck(hipMalloc((void **)&d_rawim0, sizeof(uint8_t)*size));
    gpuErrCheck(hipMalloc((void **)&d_rawim1, sizeof(uint8_t)*size));
    gpuErrCheck(hipMalloc((void **)&d_noisyim0, sizeof(uint8_t)*size));
    gpuErrCheck(hipMalloc((void **)&d_noisyim1, sizeof(uint8_t)*size));
    if (!rectified) {
        gpuErrCheck(hipMalloc((void **)&d_recim0, sizeof(uint8_t)*size));
        gpuErrCheck(hipMalloc((void **)&d_recim1, sizeof(uint8_t)*size));   
    }

    gpuErrCheck(hipMalloc((void **)&d_census0, sizeof(uint32_t)*size));
    gpuErrCheck(hipMalloc((void **)&d_census1, sizeof(uint32_t)*size));

    if (bfWidth * bfHeight != 1) {
        gpuErrCheck(hipMalloc((void **)&d_rawcost, sizeof(cost_t)*size3d));
        gpuErrCheck(hipMalloc((void **)&d_hsum, sizeof(cost_t)*size3d));
    }
    gpuErrCheck(hipMalloc((void **)&d_cost, sizeof(cost_t)*size3d));

    gpuErrCheck(hipMalloc((void **)&d_L0, sizeof(cost_t)*size3d));
    gpuErrCheck(hipMalloc((void **)&d_L1, sizeof(cost_t)*size3d));
    gpuErrCheck(hipMalloc((void **)&d_L2, sizeof(cost_t)*size3d));
    gpuErrCheck(hipMalloc((void **)&d_LAll, sizeof(cost_t)*size3d));

    gpuErrCheck(hipMalloc((void **)&d_leftDisp, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_rightDisp, sizeof(uint16_t)*size));

    if (mfSize != 1) {
        gpuErrCheck(hipMalloc((void **)&d_filteredDisp, sizeof(float)*size));
    }

#ifdef DISP_ONLY
    h_disp = new float[size];
#else
    gpuErrCheck(hipMalloc((void **)&d_a1, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_a2, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_a3, sizeof(float)*size));
    gpuErrCheck(hipMemcpyAsync(d_a1, a1.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_a2, a2.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_a3, a3.data(), sizeof(float)*size, hipMemcpyHostToDevice));

    gpuErrCheck(hipMalloc((void **)&d_depth, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_rgbDepth, sizeof(float)*rgbSize));
    h_depth = new float[rgbSize];
#endif

    gpuErrCheck(hipDeviceSynchronize());
}

py::array_t<float> DepthSensorEngine::compute(py::array_t<uint8_t> left_ndarray, py::array_t<uint8_t> right_ndarray) {
    // Convert from ndarray to Mat2d
    Mat2d<uint8_t> left = ndarray2Mat2d<uint8_t>(left_ndarray);
    Mat2d<uint8_t> right = ndarray2Mat2d<uint8_t>(right_ndarray);

    // Instance check
    if (left.rows() != right.rows() || left.cols() != right.cols()) { throw std::runtime_error("Both images must have the same size"); }
    if (cols != left.cols() || rows != left.rows()) { throw std::runtime_error("Input image size different from initiated"); }

    // Upload to GPU
    gpuErrCheck(hipMemcpyAsync(d_rawim0, left.data(), sizeof(uint8_t)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_rawim1, right.data(), sizeof(uint8_t)*size, hipMemcpyHostToDevice));

#ifdef PRINT_RUNTIME
    float runtime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif

    uint8_t *d_srcim0 = d_rawim0;
    uint8_t *d_srcim1 = d_rawim1;
    // Infrared Noise Simulation
    if (speckleShape > 0) {
#ifdef PRINT_RUNTIME
        hipEventRecord(start);
#endif
        simInfraredNoise<<<(size+WARP_SIZE-1)/(WARP_SIZE), WARP_SIZE, 0, stream1>>>(d_srcim0, d_noisyim0, d_irNoiseStates0, rows, cols, speckleShape, speckleScale, gaussianMu, gaussianSigma);
        simInfraredNoise<<<(size+WARP_SIZE-1)/(WARP_SIZE), WARP_SIZE, 0, stream2>>>(d_srcim1, d_noisyim1, d_irNoiseStates1, rows, cols, speckleShape, speckleScale, gaussianMu, gaussianSigma);
#ifdef PRINT_RUNTIME
        hipEventRecord(stop);
        gpuErrCheck(hipDeviceSynchronize());
        hipEventElapsedTime(&runtime, start, stop);
        printf("Runtime of IR noise simulation: %f ms\n", runtime);
#endif
        d_srcim0 = d_noisyim0;
        d_srcim1 = d_noisyim1;
    }

    // Rectification
    if (!rectified) {
        gpuErrCheck(hipDeviceSynchronize());
#ifdef PRINT_RUNTIME
        hipEventRecord(start);
#endif
        remap<<<(size+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(d_mapLx, d_mapLy, d_srcim0, d_recim0, rows, cols);
        remap<<<(size+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream2>>>(d_mapRx, d_mapRy, d_srcim1, d_recim1, rows, cols);
#ifdef PRINT_RUNTIME
        hipEventRecord(stop);
        gpuErrCheck(hipDeviceSynchronize());
        hipEventElapsedTime(&runtime, start, stop);
        printf("Runtime of rectification: %f ms\n", runtime);
#endif
        d_srcim0 = d_recim0;
        d_srcim1 = d_recim1;
    }

    // Center-symmetric census transform
    dim3 CSCTBlockSize;
    CSCTBlockSize.x = WARP_SIZE;
    CSCTBlockSize.y = WARP_SIZE;
    dim3 CSCTGridSize;
    CSCTGridSize.x = (cols+CSCTBlockSize.x-1) / CSCTBlockSize.x;
    CSCTGridSize.y = (rows+CSCTBlockSize.y-1) / CSCTBlockSize.y;
    int CSCTWinCols = (WARP_SIZE+censusWidth-1);
    int CSCTWinRows = (WARP_SIZE+censusHeight-1);
    int CSCTSharedMemSize = 2*CSCTWinCols*CSCTWinRows*sizeof(uint8_t);
    gpuErrCheck(hipDeviceSynchronize());
#ifdef PRINT_RUNTIME
    hipEventRecord(start);
#endif
    CSCT<<<CSCTGridSize, CSCTBlockSize, CSCTSharedMemSize, stream1>>>(d_srcim0, d_srcim1, d_census0, d_census1, rows, cols, censusWidth, censusHeight);
#ifdef PRINT_RUNTIME
    hipEventRecord(stop);
    gpuErrCheck(hipDeviceSynchronize());
    hipEventElapsedTime(&runtime, start, stop);
    printf("Runtime of CSCT: %f ms\n", runtime);
#endif

    // Hamming distance
    dim3 costGridSize;
    costGridSize.x = (cols+maxDisp-1) / maxDisp;
    costGridSize.y = rows;
    gpuErrCheck(hipDeviceSynchronize());
#ifdef PRINT_RUNTIME
    hipEventRecord(start);
#endif
    if (bfWidth * bfHeight == 1) {
        hammingCost<<<costGridSize, maxDisp, 3*maxDisp*sizeof(uint32_t), stream1>>>(d_census0, d_census1, d_cost, rows, cols, maxDisp);
    } else {
        // Apply box filter
        hammingCost<<<costGridSize, maxDisp, 3*maxDisp*sizeof(uint32_t), stream1>>>(d_census0, d_census1, d_rawcost, rows, cols, maxDisp);
        gpuErrCheck(hipDeviceSynchronize());
        boxFilterHorizontal<<<rows, maxDisp, 0, stream1>>>(d_rawcost, d_hsum, rows, cols, maxDisp, bfWidth);
        gpuErrCheck(hipDeviceSynchronize());
        boxFilterVertical<<<cols, maxDisp, 0, stream1>>>(d_hsum, d_cost, rows, cols, maxDisp, bfHeight);
    }
#ifdef PRINT_RUNTIME
    hipEventRecord(stop);
    gpuErrCheck(hipDeviceSynchronize());
    hipEventElapsedTime(&runtime, start, stop);
    printf("Runtime of cost calculation + box filter: %f ms\n", runtime);
#endif

    // Cost aggregation
    gpuErrCheck(hipDeviceSynchronize());
    int P1 = p1 * bfWidth * bfHeight;
    int P2 = p2 * bfWidth * bfHeight;
    aggrLeft2Right<<<rows, maxDisp, maxDisp*sizeof(cost_t), stream1>>>(d_cost, d_L0, P1, P2, rows, cols, maxDisp);
    aggrRight2Left<<<rows, maxDisp, maxDisp*sizeof(cost_t), stream2>>>(d_cost, d_L1, P1, P2, rows, cols, maxDisp);
    aggrTop2Bottom<<<cols, maxDisp, maxDisp*sizeof(cost_t), stream3>>>(d_cost, d_L2, P1, P2, rows, cols, maxDisp);
    gpuErrCheck(hipDeviceSynchronize());
#ifdef PRINT_RUNTIME
    hipEventRecord(start);
#endif
    aggrBottom2Top<<<cols, maxDisp, maxDisp*sizeof(cost_t), stream1>>>(d_cost, d_LAll, d_L0, d_L1, d_L2, P1, P2, rows, cols, maxDisp);
#ifdef PRINT_RUNTIME
    hipEventRecord(stop);
    gpuErrCheck(hipDeviceSynchronize());
    hipEventElapsedTime(&runtime, start, stop);
    printf("Runtime of cost aggregation: %f ms\n", runtime);
#endif

    // Winner takes all
    dim3 WTAGridSize;
    WTAGridSize.x = cols;
    WTAGridSize.y = rows;
    int WTAThrNum = ((maxDisp+WARP_SIZE-1)/WARP_SIZE) * WARP_SIZE;
    gpuErrCheck(hipDeviceSynchronize());
#ifdef PRINT_RUNTIME
    hipEventRecord(start);
#endif
    winnerTakesAll<<<WTAGridSize, WTAThrNum, maxDisp*sizeof(cost_t), stream1>>>(d_LAll, d_leftDisp, d_rightDisp, cols, maxDisp, uniqRatio);
#ifdef PRINT_RUNTIME
    hipEventRecord(stop);
    gpuErrCheck(hipDeviceSynchronize());
    hipEventElapsedTime(&runtime, start, stop);
    printf("Runtime of winner takes all: %f ms\n", runtime);
#endif

    if (lrMaxDiff != 255) {
        gpuErrCheck(hipDeviceSynchronize());
#ifdef PRINT_RUNTIME
        hipEventRecord(start);
#endif
        lrConsistencyCheck<<<(size+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(d_leftDisp, d_rightDisp, rows, cols, lrMaxDiff);
#ifdef PRINT_RUNTIME
        hipEventRecord(stop);
        gpuErrCheck(hipDeviceSynchronize());
        hipEventElapsedTime(&runtime, start, stop);
        printf("Runtime of left-right consistency check: %f ms\n", runtime);
#endif
    }

    float *d_finalDisp = d_leftDisp;
    // Apply median filter
    if (mfSize != 1) {
        int mfSharedMemSize = 2*WARP_SIZE * mfSize*mfSize * sizeof(float);
        gpuErrCheck(hipDeviceSynchronize());
#ifdef PRINT_RUNTIME
    hipEventRecord(start);
#endif
        medianFilter<<<(size+2*WARP_SIZE-1)/(2*WARP_SIZE), 2*WARP_SIZE, mfSharedMemSize, stream1>>>(d_finalDisp, d_filteredDisp, rows, cols, mfSize);
#ifdef PRINT_RUNTIME
    hipEventRecord(stop);
    gpuErrCheck(hipDeviceSynchronize());
    hipEventElapsedTime(&runtime, start, stop);
    printf("Runtime of median filter: %f ms\n", runtime);
#endif
        d_finalDisp = d_filteredDisp;
    }

#ifdef DISP_ONLY
    gpuErrCheck(hipDeviceSynchronize());
    gpuErrCheck(hipMemcpy(h_disp, d_finalDisp, sizeof(float)*size, hipMemcpyDeviceToHost));
    Mat2d<float> disp(rows, cols, h_disp);
    
    // Convert from C++ to python
    py::array_t<float> disp_ndarray = Mat2d2ndarray<float>(disp);
    return disp_ndarray;
#else
    // Convert disparity into depth
    gpuErrCheck(hipDeviceSynchronize());
    disp2Depth<<<(size+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(d_finalDisp, d_depth, size, focalLen, baselineLen);

    if (registration) {
        // Transfrom the depth map from left camera's frame to RGB camera's frame
        gpuErrCheck(hipDeviceSynchronize());
#ifdef PRINT_RUNTIME
        hipEventRecord(start);
#endif
        initRgbDepth<<<(rgbSize+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(d_rgbDepth, rgbSize, maxDepth);
        depthRegistration<<<(size+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(
            d_rgbDepth, d_depth, d_a1, d_a2, d_a3, b1, b2, b3, size, rgbRows, rgbCols);
        if (dilation) {
            depthDilation<<<(rgbSize+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(d_rgbDepth, rgbRows, rgbCols, maxDepth);
        }
        correctDepthRange<<<(rgbSize+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(d_rgbDepth, rgbSize, minDepth, maxDepth);
#ifdef PRINT_RUNTIME
        hipEventRecord(stop);
        gpuErrCheck(hipDeviceSynchronize());
        hipEventElapsedTime(&runtime, start, stop);
        printf("Runtime of registration: %f ms\n", runtime);
#endif
        
        // GPU to CPU transfer
        gpuErrCheck(hipDeviceSynchronize());
        gpuErrCheck(hipMemcpy(h_depth, d_rgbDepth, sizeof(float)*rgbSize, hipMemcpyDeviceToHost));
        Mat2d<float> depth(rgbRows, rgbCols, h_depth);
        
        // Convert from Mat2d to ndarray
        py::array_t<float> depth_ndarray = Mat2d2ndarray<float>(depth);
        return depth_ndarray;
    } else {
        correctDepthRange<<<(size+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(d_depth, size, minDepth, maxDepth);
        
        // GPU to CPU transfer
        gpuErrCheck(hipDeviceSynchronize());
        gpuErrCheck(hipMemcpy(h_depth, d_depth, sizeof(float)*size, hipMemcpyDeviceToHost));
        Mat2d<float> depth(rows, cols, h_depth);

        // Convert from Mat2d to ndarray
        py::array_t<float> depth_ndarray = Mat2d2ndarray<float>(depth);
        return depth_ndarray;
    }
#endif
}

void DepthSensorEngine::setInfraredNoiseParameters(float _speckleShape, float _speckleScale, float _gaussianMu, float _gaussianSigma) {
    speckleShape = _speckleShape;
    speckleScale = _speckleScale;
    gaussianMu = _gaussianMu;
    gaussianSigma = _gaussianSigma;
}

void DepthSensorEngine::setPenalties(uint8_t _p1, uint8_t _p2) {
    p1 = _p1;
    p2 = _p2;
}

void DepthSensorEngine::setCensusWindowSize(uint8_t _censusWidth, uint8_t _censusHeight) {
    censusWidth = _censusWidth;
    censusHeight = _censusHeight;
}

void DepthSensorEngine::setMatchingBlockSize(uint8_t _bfWidth, uint8_t _bfHeight) {
    bfWidth = _bfWidth;
    bfHeight = _bfHeight;
}

void DepthSensorEngine::setUniquenessRatio(uint8_t _uniqRatio) {
    uniqRatio = _uniqRatio;
}

void DepthSensorEngine::setLrMaxDiff(uint8_t _lrMaxDiff) {
    lrMaxDiff = _lrMaxDiff;
}

DepthSensorEngine::~DepthSensorEngine() {
    gpuErrCheck(hipStreamDestroy(stream1));
    gpuErrCheck(hipStreamDestroy(stream2));
    gpuErrCheck(hipStreamDestroy(stream3));

    gpuErrCheck(hipFree(d_irNoiseStates0));
    gpuErrCheck(hipFree(d_irNoiseStates1));

    gpuErrCheck(hipFree(d_mapLx));
    gpuErrCheck(hipFree(d_mapLy));
    gpuErrCheck(hipFree(d_mapRx));
    gpuErrCheck(hipFree(d_mapRy));

    gpuErrCheck(hipFree(d_rawim0));
    gpuErrCheck(hipFree(d_rawim1));
    gpuErrCheck(hipFree(d_noisyim0));
    gpuErrCheck(hipFree(d_noisyim1));
    if (!rectified) {
        gpuErrCheck(hipFree(d_recim0));
        gpuErrCheck(hipFree(d_recim1));
    }

    gpuErrCheck(hipFree(d_census0));
    gpuErrCheck(hipFree(d_census1));
    
    if (bfWidth * bfHeight != 1) {
        gpuErrCheck(hipFree(d_rawcost));
        gpuErrCheck(hipFree(d_hsum));
    }
    gpuErrCheck(hipFree(d_cost));

    gpuErrCheck(hipFree(d_L0));
    gpuErrCheck(hipFree(d_L1));
    gpuErrCheck(hipFree(d_L2));
    gpuErrCheck(hipFree(d_LAll));

    gpuErrCheck(hipFree(d_leftDisp));
    gpuErrCheck(hipFree(d_rightDisp));

    if (mfSize != 1) {
        gpuErrCheck(hipFree(d_filteredDisp));
    }

#ifdef DISP_ONLY
    delete h_disp;
#else
    gpuErrCheck(hipFree(d_depth));
    if (registration) {
        gpuErrCheck(hipFree(d_a1));
        gpuErrCheck(hipFree(d_a2));
        gpuErrCheck(hipFree(d_a3));
        gpuErrCheck(hipFree(d_rgbDepth));
    }
    delete h_depth;
#endif

    gpuErrCheck(hipDeviceSynchronize());
}

}
