#include "core.h"

using namespace simsense;

namespace simsense {

// Constructor without registration
DepthSensorEngine::DepthSensorEngine(
        uint32_t _rows, uint32_t _cols, float _focalLen, float _baselineLen, float _minDepth, float _maxDepth, bool _rectified,
        uint8_t _censusWidth, uint8_t _censusHeight, uint32_t _maxDisp, uint8_t _bfWidth, uint8_t _bfHeight, uint8_t _p1, uint8_t _p2,
        uint8_t _uniqRatio, uint8_t _lrMaxDiff, uint8_t _mfSize, py::array_t<float> map_lx, py::array_t<float> map_ly,
        py::array_t<float> map_rx, py::array_t<float> map_ry
    ) {
    // Convert from python to C++
    Mat2d<float> mapLx = ndarray2Mat2d<float>(map_lx);
    Mat2d<float> mapLy = ndarray2Mat2d<float>(map_ly);
    Mat2d<float> mapRx = ndarray2Mat2d<float>(map_rx);
    Mat2d<float> mapRy = ndarray2Mat2d<float>(map_ry);

    // Create streams and free memory if necessary
    gpuErrCheck(hipStreamCreate(&stream1));
    gpuErrCheck(hipStreamCreate(&stream2));
    gpuErrCheck(hipStreamCreate(&stream3));

    // Intialize class variables
    censusWidth = _censusWidth;
    censusHeight = _censusHeight;
    bfWidth = _bfWidth;
    bfHeight = _bfHeight;
    p1 = _p1 * bfWidth * bfHeight;
    p2 = _p2 * bfWidth * bfHeight;
    uniqRatio = _uniqRatio;
    lrMaxDiff = _lrMaxDiff;
    mfSize = _mfSize;
    rows = _rows;
    cols = _cols;
    size = rows*cols;
    maxDisp = _maxDisp;
    focalLen = _focalLen;
    baselineLen = _baselineLen;
    minDepth = _minDepth;
    maxDepth = _maxDepth;
    rectified = _rectified;
    registration = false;
    int size3d = size*maxDisp;

    // Allocate GPU memory
    gpuErrCheck(hipMalloc((void **)&d_mapLx, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_mapLy, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_mapRx, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_mapRy, sizeof(float)*size));
    gpuErrCheck(hipMemcpyAsync(d_mapLx, mapLx.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_mapLy, mapLy.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_mapRx, mapRx.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_mapRy, mapRy.data(), sizeof(float)*size, hipMemcpyHostToDevice));

    if (!rectified) {
        gpuErrCheck(hipMalloc((void **)&d_rawim0, sizeof(uint8_t)*size));
        gpuErrCheck(hipMalloc((void **)&d_rawim1, sizeof(uint8_t)*size));
    }
    gpuErrCheck(hipMalloc((void **)&d_im0, sizeof(uint8_t)*size));
    gpuErrCheck(hipMalloc((void **)&d_im1, sizeof(uint8_t)*size));

    gpuErrCheck(hipMalloc((void **)&d_census0, sizeof(uint32_t)*size));
    gpuErrCheck(hipMalloc((void **)&d_census1, sizeof(uint32_t)*size));

    if (bfWidth * bfHeight != 1) {
        gpuErrCheck(hipMalloc((void **)&d_rawcost, sizeof(cost_t)*size3d));
        gpuErrCheck(hipMalloc((void **)&d_hsum, sizeof(cost_t)*size3d));
    }
    gpuErrCheck(hipMalloc((void **)&d_cost, sizeof(cost_t)*size3d));

    gpuErrCheck(hipMalloc((void **)&d_L0, sizeof(cost_t)*size3d));
    gpuErrCheck(hipMalloc((void **)&d_L1, sizeof(cost_t)*size3d));
    gpuErrCheck(hipMalloc((void **)&d_L2, sizeof(cost_t)*size3d));
    gpuErrCheck(hipMalloc((void **)&d_LAll, sizeof(cost_t)*size3d));

    gpuErrCheck(hipMalloc((void **)&d_leftDisp, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_rightDisp, sizeof(uint16_t)*size));

    if (mfSize != 1) {
        gpuErrCheck(hipMalloc((void **)&d_filteredDisp, sizeof(float)*size));
    }

#ifndef DISP_ONLY
    gpuErrCheck(hipMalloc((void **)&d_depth, sizeof(float)*size));
#endif

    gpuErrCheck(hipDeviceSynchronize());
}

// Constructor with registration
DepthSensorEngine::DepthSensorEngine(
        uint32_t _rows, uint32_t _cols, uint32_t _rgbRows, uint32_t _rgbCols, float _focalLen, float _baselineLen, float _minDepth,
        float _maxDepth, bool _rectified, uint8_t _censusWidth, uint8_t _censusHeight, uint32_t _maxDisp, uint8_t _bfWidth, uint8_t _bfHeight,
        uint8_t _p1, uint8_t _p2, uint8_t _uniqRatio, uint8_t _lrMaxDiff, uint8_t _mfSize, py::array_t<float> map_lx, py::array_t<float> map_ly,
        py::array_t<float> map_rx, py::array_t<float> map_ry, py::array_t<float> _a1, py::array_t<float> _a2, py::array_t<float> _a3,
        float _b1, float _b2, float _b3, bool _dilation
    ) {
    // Convert from python to C++
    Mat2d<float> mapLx = ndarray2Mat2d<float>(map_lx);
    Mat2d<float> mapLy = ndarray2Mat2d<float>(map_ly);
    Mat2d<float> mapRx = ndarray2Mat2d<float>(map_rx);
    Mat2d<float> mapRy = ndarray2Mat2d<float>(map_ry);
    Mat2d<float> a1 = ndarray2Mat2d<float>(_a1);
    Mat2d<float> a2 = ndarray2Mat2d<float>(_a2);
    Mat2d<float> a3 = ndarray2Mat2d<float>(_a3);

    // Create streams and free memory if necessary
    gpuErrCheck(hipStreamCreate(&stream1));
    gpuErrCheck(hipStreamCreate(&stream2));
    gpuErrCheck(hipStreamCreate(&stream3));

    // Intialize class variables
    censusWidth = _censusWidth;
    censusHeight = _censusHeight;
    bfWidth = _bfWidth;
    bfHeight = _bfHeight;
    p1 = _p1 * bfWidth * bfHeight;
    p2 = _p2 * bfWidth * bfHeight;
    uniqRatio = _uniqRatio;
    lrMaxDiff = _lrMaxDiff;
    mfSize = _mfSize;
    rows = _rows;
    cols = _cols;
    size = rows*cols;
    maxDisp = _maxDisp;
    int size3d = size*maxDisp;
    focalLen = _focalLen;
    baselineLen = _baselineLen;
    minDepth = _minDepth;
    maxDepth = _maxDepth;
    rectified = _rectified;
    registration = true;
    dilation = _dilation;
    rgbRows = _rgbRows;
    rgbCols = _rgbCols;
    rgbSize = rgbRows*rgbCols;
    b1 = _b1;
    b2 = _b2;
    b3 = _b3;

    // Allocate GPU memory
    gpuErrCheck(hipMalloc((void **)&d_mapLx, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_mapLy, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_mapRx, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_mapRy, sizeof(float)*size));
    gpuErrCheck(hipMemcpyAsync(d_mapLx, mapLx.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_mapLy, mapLy.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_mapRx, mapRx.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_mapRy, mapRy.data(), sizeof(float)*size, hipMemcpyHostToDevice));

    if (!rectified) {
        gpuErrCheck(hipMalloc((void **)&d_rawim0, sizeof(uint8_t)*size));
        gpuErrCheck(hipMalloc((void **)&d_rawim1, sizeof(uint8_t)*size));
    }
    gpuErrCheck(hipMalloc((void **)&d_im0, sizeof(uint8_t)*size));
    gpuErrCheck(hipMalloc((void **)&d_im1, sizeof(uint8_t)*size));

    gpuErrCheck(hipMalloc((void **)&d_census0, sizeof(uint32_t)*size));
    gpuErrCheck(hipMalloc((void **)&d_census1, sizeof(uint32_t)*size));

    if (bfWidth * bfHeight != 1) {
        gpuErrCheck(hipMalloc((void **)&d_rawcost, sizeof(cost_t)*size3d));
        gpuErrCheck(hipMalloc((void **)&d_hsum, sizeof(cost_t)*size3d));
    }
    gpuErrCheck(hipMalloc((void **)&d_cost, sizeof(cost_t)*size3d));

    gpuErrCheck(hipMalloc((void **)&d_L0, sizeof(cost_t)*size3d));
    gpuErrCheck(hipMalloc((void **)&d_L1, sizeof(cost_t)*size3d));
    gpuErrCheck(hipMalloc((void **)&d_L2, sizeof(cost_t)*size3d));
    gpuErrCheck(hipMalloc((void **)&d_LAll, sizeof(cost_t)*size3d));

    gpuErrCheck(hipMalloc((void **)&d_leftDisp, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_rightDisp, sizeof(uint16_t)*size));

    if (mfSize != 1) {
        gpuErrCheck(hipMalloc((void **)&d_filteredDisp, sizeof(float)*size));
    }

#ifndef DISP_ONLY
    gpuErrCheck(hipMalloc((void **)&d_a1, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_a2, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_a3, sizeof(float)*size));
    gpuErrCheck(hipMemcpyAsync(d_a1, a1.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_a2, a2.data(), sizeof(float)*size, hipMemcpyHostToDevice));
    gpuErrCheck(hipMemcpyAsync(d_a3, a3.data(), sizeof(float)*size, hipMemcpyHostToDevice));

    gpuErrCheck(hipMalloc((void **)&d_depth, sizeof(float)*size));
    gpuErrCheck(hipMalloc((void **)&d_rgbDepth, sizeof(float)*rgbSize));
#endif

    gpuErrCheck(hipDeviceSynchronize());
}

py::array_t<float> DepthSensorEngine::compute(py::array_t<uint8_t> left_ndarray, py::array_t<uint8_t> right_ndarray) {
    // Convert from python to C++
    Mat2d<uint8_t> left = ndarray2Mat2d<uint8_t>(left_ndarray);
    Mat2d<uint8_t> right = ndarray2Mat2d<uint8_t>(right_ndarray);

    if (left.rows() != right.rows() || left.cols() != right.cols()) { throw std::runtime_error("Both images must have the same size"); }
    if (cols != left.cols() || rows != left.rows()) { throw std::runtime_error("Input image size different from initiated"); }

#ifdef PRINT_RUNTIME
    float runtime;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
#endif
    
    if (rectified) {
        gpuErrCheck(hipMemcpyAsync(d_im0, left.data(), sizeof(uint8_t)*size, hipMemcpyHostToDevice));
        gpuErrCheck(hipMemcpyAsync(d_im1, right.data(), sizeof(uint8_t)*size, hipMemcpyHostToDevice));
    } else {
        gpuErrCheck(hipMemcpyAsync(d_rawim0, left.data(), sizeof(uint8_t)*size, hipMemcpyHostToDevice));
        gpuErrCheck(hipMemcpyAsync(d_rawim1, right.data(), sizeof(uint8_t)*size, hipMemcpyHostToDevice));
        gpuErrCheck(hipDeviceSynchronize());

#ifdef PRINT_RUNTIME
        hipEventRecord(start);
#endif
        // Rectification
        remap<<<(size+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(d_mapLx, d_mapLy, d_rawim0, d_im0, rows, cols);
        remap<<<(size+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream2>>>(d_mapRx, d_mapRy, d_rawim1, d_im1, rows, cols);
#ifdef PRINT_RUNTIME
        hipEventRecord(stop);
        gpuErrCheck(hipDeviceSynchronize());
        hipEventElapsedTime(&runtime, start, stop);
        printf("Runtime of rectification: %f ms\n", runtime);
#endif
    }

#ifdef PRINT_RUNTIME
    hipEventRecord(start);
#endif
    // Center-symmetric census transform
    dim3 CSCTBlockSize;
    CSCTBlockSize.x = WARP_SIZE;
    CSCTBlockSize.y = WARP_SIZE;
    dim3 CSCTGridSize;
    CSCTGridSize.x = (cols+CSCTBlockSize.x-1) / CSCTBlockSize.x;
    CSCTGridSize.y = (rows+CSCTBlockSize.y-1) / CSCTBlockSize.y;
    int CSCTWinCols = (WARP_SIZE+censusWidth-1);
    int CSCTWinRows = (WARP_SIZE+censusHeight-1);
    int CSCTSharedMemSize = 2*CSCTWinCols*CSCTWinRows*sizeof(uint8_t);
    gpuErrCheck(hipDeviceSynchronize());
    CSCT<<<CSCTGridSize, CSCTBlockSize, CSCTSharedMemSize, stream1>>>(d_im0, d_im1, d_census0, d_census1, rows, cols, censusWidth, censusHeight);
#ifdef PRINT_RUNTIME
    hipEventRecord(stop);
    gpuErrCheck(hipDeviceSynchronize());
    hipEventElapsedTime(&runtime, start, stop);
    printf("Runtime of CSCT: %f ms\n", runtime);
#endif

#ifdef PRINT_RUNTIME
    hipEventRecord(start);
#endif
    // Hamming distance
    dim3 costGridSize;
    costGridSize.x = (cols+maxDisp-1) / maxDisp;
    costGridSize.y = rows;
    gpuErrCheck(hipDeviceSynchronize());
    if (bfWidth * bfHeight == 1) {
        hammingCost<<<costGridSize, maxDisp, 3*maxDisp*sizeof(uint32_t), stream1>>>(d_census0, d_census1, d_cost, rows, cols, maxDisp);
    } else {
        // Apply box filter
        hammingCost<<<costGridSize, maxDisp, 3*maxDisp*sizeof(uint32_t), stream1>>>(d_census0, d_census1, d_rawcost, rows, cols, maxDisp);
        gpuErrCheck(hipDeviceSynchronize());
        boxFilterHorizontal<<<rows, maxDisp, 0, stream1>>>(d_rawcost, d_hsum, rows, cols, maxDisp, bfWidth);
        gpuErrCheck(hipDeviceSynchronize());
        boxFilterVertical<<<cols, maxDisp, 0, stream1>>>(d_hsum, d_cost, rows, cols, maxDisp, bfHeight);
    }
#ifdef PRINT_RUNTIME
    hipEventRecord(stop);
    gpuErrCheck(hipDeviceSynchronize());
    hipEventElapsedTime(&runtime, start, stop);
    printf("Runtime of cost calculation + box filter: %f ms\n", runtime);
#endif

#ifdef PRINT_RUNTIME
    hipEventRecord(start);
#endif
    // Cost aggregation
    gpuErrCheck(hipDeviceSynchronize());
    aggrLeft2Right<<<rows, maxDisp, maxDisp*sizeof(cost_t), stream1>>>(d_cost, d_L0, p1, p2, rows, cols, maxDisp);
    aggrRight2Left<<<rows, maxDisp, maxDisp*sizeof(cost_t), stream2>>>(d_cost, d_L1, p1, p2, rows, cols, maxDisp);
    aggrTop2Bottom<<<cols, maxDisp, maxDisp*sizeof(cost_t), stream3>>>(d_cost, d_L2, p1, p2, rows, cols, maxDisp);
    gpuErrCheck(hipDeviceSynchronize());
    aggrBottom2Top<<<cols, maxDisp, maxDisp*sizeof(cost_t), stream1>>>(d_cost, d_LAll, d_L0, d_L1, d_L2, p1, p2, rows, cols, maxDisp);
#ifdef PRINT_RUNTIME
    hipEventRecord(stop);
    gpuErrCheck(hipDeviceSynchronize());
    hipEventElapsedTime(&runtime, start, stop);
    printf("Runtime of cost aggregation: %f ms\n", runtime);
#endif

#ifdef PRINT_RUNTIME
    hipEventRecord(start);
#endif
    // Winner takes all
    dim3 WTAGridSize;
    WTAGridSize.x = cols;
    WTAGridSize.y = rows;
    int WTAThrNum = ((maxDisp+WARP_SIZE-1)/WARP_SIZE) * WARP_SIZE;
    gpuErrCheck(hipDeviceSynchronize());
    winnerTakesAll<<<WTAGridSize, WTAThrNum, maxDisp*sizeof(cost_t), stream1>>>(d_LAll, d_leftDisp, d_rightDisp, cols, maxDisp, uniqRatio);
#ifdef PRINT_RUNTIME
    hipEventRecord(stop);
    gpuErrCheck(hipDeviceSynchronize());
    hipEventElapsedTime(&runtime, start, stop);
    printf("Runtime of winner takes all: %f ms\n", runtime);
#endif

    if (lrMaxDiff != 255) {
#ifdef PRINT_RUNTIME
        hipEventRecord(start);
#endif
        gpuErrCheck(hipDeviceSynchronize());
        lrConsistencyCheck<<<(size+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(d_leftDisp, d_rightDisp, rows, cols, lrMaxDiff);
#ifdef PRINT_RUNTIME
        hipEventRecord(stop);
        gpuErrCheck(hipDeviceSynchronize());
        hipEventElapsedTime(&runtime, start, stop);
        printf("Runtime of left-right consistency check: %f ms\n", runtime);
#endif
    }

    float *d_finalDisp = d_leftDisp;
#ifdef PRINT_RUNTIME
    hipEventRecord(start);
#endif
    // Apply median filter
    if (mfSize != 1) {
        int mfSharedMemSize = 2*WARP_SIZE * mfSize*mfSize * sizeof(float);
        gpuErrCheck(hipDeviceSynchronize());
        medianFilter<<<(size+2*WARP_SIZE-1)/(2*WARP_SIZE), 2*WARP_SIZE, mfSharedMemSize, stream1>>>(d_finalDisp, d_filteredDisp, rows, cols, mfSize);
        d_finalDisp = d_filteredDisp;
    }
#ifdef PRINT_RUNTIME
    hipEventRecord(stop);
    gpuErrCheck(hipDeviceSynchronize());
    hipEventElapsedTime(&runtime, start, stop);
    printf("Runtime of median filter: %f ms\n", runtime);
#endif

#ifdef DISP_ONLY
    float *h_disp = new float[size];
    gpuErrCheck(hipDeviceSynchronize());
    gpuErrCheck(hipMemcpy(h_disp, d_finalDisp, sizeof(float)*size, hipMemcpyDeviceToHost));
    Mat2d<float> disp(rows, cols, h_disp);
    
    // Convert from C++ to python
    py::array_t<float> disp_ndarray = Mat2d2ndarray<float>(disp);
    return disp_ndarray;
#else
    // Convert disparity into depth
    gpuErrCheck(hipDeviceSynchronize());
    disp2Depth<<<(size+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(d_finalDisp, d_depth, size, focalLen, baselineLen);

    if (registration) {
        // Transfrom the depth map from left camera's frame to RGB camera's frame
#ifdef PRINT_RUNTIME
        hipEventRecord(start);
#endif
        gpuErrCheck(hipDeviceSynchronize());
        initRgbDepth<<<(rgbSize+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(d_rgbDepth, rgbSize, maxDepth);
        depthRegistration<<<(size+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(
            d_rgbDepth, d_depth, d_a1, d_a2, d_a3, b1, b2, b3, size, rgbRows, rgbCols);
        if (dilation) {
            depthDilation<<<(rgbSize+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(d_rgbDepth, rgbRows, rgbCols, maxDepth);
        }
        correctDepthRange<<<(rgbSize+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(d_rgbDepth, rgbSize, minDepth, maxDepth);
#ifdef PRINT_RUNTIME
        hipEventRecord(stop);
        gpuErrCheck(hipDeviceSynchronize());
        hipEventElapsedTime(&runtime, start, stop);
        printf("Runtime of registration: %f ms\n", runtime);
#endif
        
        // GPU to CPU transfer
        float *h_depth = new float[rgbSize];
        gpuErrCheck(hipDeviceSynchronize());
        gpuErrCheck(hipMemcpy(h_depth, d_rgbDepth, sizeof(float)*rgbSize, hipMemcpyDeviceToHost));
        Mat2d<float> depth(rgbRows, rgbCols, h_depth);
        
        // Convert from C++ to python
        py::array_t<float> depth_ndarray = Mat2d2ndarray<float>(depth);
        return depth_ndarray;
    } else {
        correctDepthRange<<<(size+8*WARP_SIZE-1)/(8*WARP_SIZE), 8*WARP_SIZE, 0, stream1>>>(d_depth, size, minDepth, maxDepth);
        
        // GPU to CPU transfer
        float *h_depth = new float[size];
        gpuErrCheck(hipDeviceSynchronize());
        gpuErrCheck(hipMemcpy(h_depth, d_depth, sizeof(float)*size, hipMemcpyDeviceToHost));
        Mat2d<float> depth(rows, cols, h_depth);

        // Convert from C++ to python
        py::array_t<float> depth_ndarray = Mat2d2ndarray<float>(depth);
        return depth_ndarray;
    }
#endif
}

DepthSensorEngine::~DepthSensorEngine() {
    gpuErrCheck(hipStreamDestroy(stream1));
    gpuErrCheck(hipStreamDestroy(stream2));
    gpuErrCheck(hipStreamDestroy(stream3));

    gpuErrCheck(hipFree(d_mapLx));
    gpuErrCheck(hipFree(d_mapLy));
    gpuErrCheck(hipFree(d_mapRx));
    gpuErrCheck(hipFree(d_mapRy));

    if (!rectified) {
        gpuErrCheck(hipFree(d_rawim0));
        gpuErrCheck(hipFree(d_rawim1));
    }
    gpuErrCheck(hipFree(d_im0));
    gpuErrCheck(hipFree(d_im1));

    gpuErrCheck(hipFree(d_census0));
    gpuErrCheck(hipFree(d_census1));
    
    if (bfWidth * bfHeight != 1) {
        gpuErrCheck(hipFree(d_rawcost));
        gpuErrCheck(hipFree(d_hsum));
    }
    gpuErrCheck(hipFree(d_cost));

    gpuErrCheck(hipFree(d_L0));
    gpuErrCheck(hipFree(d_L1));
    gpuErrCheck(hipFree(d_L2));
    gpuErrCheck(hipFree(d_LAll));

    gpuErrCheck(hipFree(d_leftDisp));
    gpuErrCheck(hipFree(d_rightDisp));

    if (mfSize != 1) {
        gpuErrCheck(hipFree(d_filteredDisp));
    }

#ifndef DISP_ONLY
    gpuErrCheck(hipFree(d_depth));
#endif

    gpuErrCheck(hipDeviceSynchronize());
}

}
